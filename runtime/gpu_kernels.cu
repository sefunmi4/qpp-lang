#include "hip/hip_runtime.h"
#include "gpu_kernels.h"
#ifdef USE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

namespace qpp {

__global__ void single_qubit_kernel(hipDoubleComplex* state, std::size_t step, std::size_t size,
                                    hipDoubleComplex m00, hipDoubleComplex m01,
                                    hipDoubleComplex m10, hipDoubleComplex m11) {
    std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t start = idx * step * 2;
    if (start >= size) return;
    for (std::size_t j = 0; j < step; ++j) {
        std::size_t pos0 = start + j;
        std::size_t pos1 = start + j + step;
        hipDoubleComplex a = state[pos0];
        hipDoubleComplex b = state[pos1];
        state[pos0] = hipCadd(hipCmul(m00, a), hipCmul(m01, b));
        state[pos1] = hipCadd(hipCmul(m10, a), hipCmul(m11, b));
    }
}

void gpu_apply_single_qubit_gate(std::vector<std::complex<double>>& st,
                                 std::size_t target,
                                 const std::complex<double> mat[2][2]) {
    std::size_t size = st.size();
    std::size_t step = 1ULL << target;
    hipDoubleComplex* d_state;
    hipMalloc(&d_state, sizeof(hipDoubleComplex)*size);
    hipMemcpy(d_state, st.data(), sizeof(hipDoubleComplex)*size, hipMemcpyHostToDevice);
    hipDoubleComplex m00 = make_hipDoubleComplex(mat[0][0].real(), mat[0][0].imag());
    hipDoubleComplex m01 = make_hipDoubleComplex(mat[0][1].real(), mat[0][1].imag());
    hipDoubleComplex m10 = make_hipDoubleComplex(mat[1][0].real(), mat[1][0].imag());
    hipDoubleComplex m11 = make_hipDoubleComplex(mat[1][1].real(), mat[1][1].imag());
    dim3 block(64);
    dim3 grid((size/(step*2)+block.x-1)/block.x);
    single_qubit_kernel<<<grid, block>>>(d_state, step, size, m00, m01, m10, m11);
    hipMemcpy(st.data(), d_state, sizeof(hipDoubleComplex)*size, hipMemcpyDeviceToHost);
    hipFree(d_state);
}

__global__ void cnot_kernel(hipDoubleComplex* state, std::size_t size, std::size_t cbit, std::size_t tbit) {
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    if ((i & cbit) && !(i & tbit)) {
        std::size_t j = i | tbit;
        hipDoubleComplex tmp = state[i];
        state[i] = state[j];
        state[j] = tmp;
    }
}

void gpu_apply_cnot(std::vector<std::complex<double>>& st,
                    std::size_t control, std::size_t target) {
    std::size_t size = st.size();
    hipDoubleComplex* d_state;
    hipMalloc(&d_state, sizeof(hipDoubleComplex)*size);
    hipMemcpy(d_state, st.data(), sizeof(hipDoubleComplex)*size, hipMemcpyHostToDevice);
    dim3 block(64);
    dim3 grid((size+block.x-1)/block.x);
    cnot_kernel<<<grid, block>>>(d_state, size, 1ULL<<control, 1ULL<<target);
    hipMemcpy(st.data(), d_state, sizeof(hipDoubleComplex)*size, hipMemcpyDeviceToHost);
    hipFree(d_state);
}

} // namespace qpp
#endif
